

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_SIZE  16
#define HEADER_SIZE 122

typedef unsigned char BYTE;

/**
 * Structure that represents a BMP image.
 */
typedef struct
{
    int   width;
    int   height;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
 * Reads a BMP 24bpp file and returns a BMPImage structure.
 * Thanks to https://stackoverflow.com/a/9296467
 */
BMPImage readBMP(char *filename)
{
    BMPImage bitmap = { 0 };
    int      size   = 0;
    BYTE     *data  = NULL;
    FILE     *file  = fopen(filename, "rb");
    
    // Read the header (expected BGR - 24bpp)
    fread(g_info, sizeof(BYTE), HEADER_SIZE, file);

    // Get the image width / height from the header
    bitmap.width  = *((int *)&g_info[18]);
    bitmap.height = *((int *)&g_info[22]);
    size          = *((int *)&g_info[34]);
    
    // Read the image data
    data = (BYTE *)malloc(sizeof(BYTE) * size);
    fread(data, sizeof(BYTE), size, file);
    
    // Convert the pixel values to float
    bitmap.data = (float *)malloc(sizeof(float) * size);
    
    for (int i = 0; i < size; i++)
    {
        bitmap.data[i] = (float)data[i];
    }
    
    fclose(file);
    free(data);
    
    return bitmap;
}

/**
 * Writes a BMP file in grayscale given its image data and a filename.
 */
void writeBMPGrayscale(int width, int height, float *image, char *filename)
{
    FILE *file = NULL;
    
    file = fopen(filename, "wb");
    
    // Write the reference header
    fwrite(g_info, sizeof(BYTE), HEADER_SIZE, file);
    
    // Unwrap the 8-bit grayscale into a 24bpp (for simplicity)
    for (int h = 0; h < height; h++)
    {
        int offset = h * width;
        
        for (int w = 0; w < width; w++)
        {
            BYTE pixel = (BYTE)((image[offset + w] > 255.0f) ? 255.0f :
                                (image[offset + w] < 0.0f)   ? 0.0f   :
                                                               image[offset + w]);
            
            // Repeat the same pixel value for BGR
            fputc(pixel, file);
            fputc(pixel, file);
            fputc(pixel, file);
        }
    }
    
    fclose(file);
}

/**
 * Releases a given BMPImage.
 */
void freeBMP(BMPImage bitmap)
{
    free(bitmap.data);
}

/**
 * Checks if there has been any CUDA error. The method will automatically print
 * some information and exit the program when an error is found.
 */
void checkCUDAError()
{
    hipError_t hipError_t = hipGetLastError();
    
    if(hipError_t != hipSuccess)
    {
        printf("CUDA Error: Returned %d: %s\n", hipError_t,
                                                hipGetErrorString(hipError_t));
        exit(-1);
    }
}

/**
 * Calculates the elapsed time between two time intervals (in milliseconds).
 */
long get_elapsed(tval t0, tval t1)
{
    return (t1.tv_sec - t0.tv_sec) * 1000 + (t1.tv_usec - t0.tv_usec) / 1000;
}

/**
 * Stores the result image and prints a message.
 */
void store_result(int index, long elapsed_cpu, long elapsed_gpu,
                  int width, int height, float *image)
{
    char path[255];
    
    sprintf(path, "images/lab02_result_%d.bmp", index);
    writeBMPGrayscale(width, height, image, path);
    
    printf("Step #%d Completed - Result stored in \"%s\".\n", index, path);
    printf("Elapsed CPU: %ldms / ", elapsed_cpu);
    
    if (elapsed_gpu == 0)
    {
        printf("[GPU version not available]\n");
    }
    else
    {
        printf("Elapsed GPU: %ldms\n", elapsed_gpu);
    }
}

/**
 * Converts a given 24bpp image into 8bpp grayscale using the CPU.
 */
void cpu_grayscale(int width, int height, float *image, float *image_out)
{
    for (int h = 0; h < height; h++)
    {
        int offset_out = h * width;      // 1 color per pixel
        int offset     = offset_out * 3; // 3 colors per pixel
        
        for (int w = 0; w < width; w++)
        {
            float *pixel = &image[offset + w * 3];
            
            // Convert to grayscale following the "luminance" model
            image_out[offset_out + w] = pixel[0] * 0.0722f + // B
                                        pixel[1] * 0.7152f + // G
                                        pixel[2] * 0.2126f;  // R
        }
    }
}

/**
 * Applies a 3x3 convolution matrix to a pixel using the CPU / GPU.
 */
float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    float pixel = 0.0f;
    
    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;
        
        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }
    
    return pixel;
}

/**
 * Applies a Gaussian 3x3 filter to a given image using the CPU.
 */
void cpu_gaussian(int width, int height, float *image, float *image_out)
{
    float gaussian[9] = { 1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f,
                          2.0f / 16.0f, 4.0f / 16.0f, 2.0f / 16.0f,
                          1.0f / 16.0f, 2.0f / 16.0f, 1.0f / 16.0f };
    
    for (int h = 0; h < (height - 2); h++)
    {
        int offset_t = h * width;
        int offset   = (h + 1) * width;
        
        for (int w = 0; w < (width - 2); w++)
        {
            image_out[offset + (w + 1)] = cpu_applyFilter(&image[offset_t + w],
                                                          width, gaussian, 3);
        }
    }
}

/**
 * Calculates the gradient of an image using a Sobel filter on the CPU.
 */
void cpu_sobel(int width, int height, float *image, float *image_out)
{
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };
    
    for (int h = 0; h < (height - 2); h++)
    {
        int offset_t = h * width;
        int offset   = (h + 1) * width;
        
        for (int w = 0; w < (width - 2); w++)
        {
            float gx = cpu_applyFilter(&image[offset_t + w], width, sobel_x, 3);
            float gy = cpu_applyFilter(&image[offset_t + w], width, sobel_y, 3);
            
            // Note: The output can be negative or exceed the max. color value
            // of 255. We compensate this afterwards while storing the file.
            image_out[offset + (w + 1)] = sqrtf(gx * gx + gy * gy);
        }
    }
}




//////////////////////////////////////
// FORTRAN <-> C INTEROP. FUNCTIONS //
//////////////////////////////////////

BMPImage bitmap;
tval     t[2];
long     elapsed[2];

/**
 * Read the input Bitmap file.
 */
extern "C" void c_readbmp_(float *image)
{
    char filename[255] = "images/lab02.bmp";
    
    bitmap = readBMP(filename);
    memcpy(image, bitmap.data, sizeof(float) * bitmap.width * bitmap.height * 3);
    
    printf("Image opened (width=%d height=%d).\n", bitmap.width, bitmap.height);
}

/**
 * Release the input Bitmap file.
 */
extern "C" void c_freebmp_()
{
    freeBMP(bitmap);
}

/**
 * Step 1: Convert to grayscale (begin)
 */
extern "C" void c_step1_begin_(float *image, float *image_out)
{
    // Launch the CPU version
    gettimeofday(&t[0], NULL);
    cpu_grayscale(bitmap.width, bitmap.height, image, image_out);
    gettimeofday(&t[1], NULL);
    
    elapsed[0] = get_elapsed(t[0], t[1]);
    
    // Launch the GPU version
    gettimeofday(&t[0], NULL);
}

/**
 * Step 1: Convert to grayscale (end)
 */
extern "C" void c_step1_end_(float *image_out)
{
    gettimeofday(&t[1], NULL);
    
    elapsed[1] = get_elapsed(t[0], t[1]);
    
    // Store the result image in grayscale
    store_result(1, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out);
}

/**
 * Step 2: Apply a 3x3 Gaussian filter (begin)
 */
extern "C" void c_step2_begin_(float *image, float *image_out)
{
    // Launch the CPU version
    gettimeofday(&t[0], NULL);
    cpu_gaussian(bitmap.width, bitmap.height, image, image_out);
    gettimeofday(&t[1], NULL);
    
    elapsed[0] = get_elapsed(t[0], t[1]);
    
    // Launch the GPU version
    gettimeofday(&t[0], NULL);
}

/**
 * Step 2: Apply a 3x3 Gaussian filter (end)
 */
extern "C" void c_step2_end_(float *image_out)
{
    gettimeofday(&t[1], NULL);
    
    elapsed[1] = get_elapsed(t[0], t[1]);
    
    // Store the result image with the Gaussian filter applied
    store_result(2, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out);
}

/**
 * Step 3: Apply a Sobel filter (begin)
 */
extern "C" void c_step3_begin_(float *image, float *image_out)
{
    // Launch the CPU version
    gettimeofday(&t[0], NULL);
    cpu_sobel(bitmap.width, bitmap.height, image, image_out);
    gettimeofday(&t[1], NULL);
    
    elapsed[0] = get_elapsed(t[0], t[1]);
    
    // Launch the GPU version
    gettimeofday(&t[0], NULL);
}

/**
 * Step 3: Apply a Sobel filter (end)
 */
extern "C" void c_step3_end_(float *image_out)
{
    gettimeofday(&t[1], NULL);
    
    elapsed[1] = get_elapsed(t[0], t[1]);
    
    // Store the final result image with the Sobel filter applied
    store_result(3, elapsed[0], elapsed[1], bitmap.width, bitmap.height, image_out);
}

